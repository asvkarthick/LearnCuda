/* Program to add two arrays in GPU using multiple blocks
   nvcc program-005.cu
 */
#include <iostream>
#include <hip/hip_runtime.h>

// Number of elements to add is 10
#define N 10

__global__ void add(int* a, int* b, int* c) {
    int index = blockIdx.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int a[N], b[N], c[N];
    int *gpuMem1, *gpuMem2, *gpuResult;

    // Allocate the GPU input and output memories
    if (hipMalloc((void**)&gpuMem1, N * sizeof(int)) != hipSuccess) {
        std::cerr << "Failed to allocate GPU mem1" << std::endl;
	return -1;
    }
    if (hipMalloc((void**)&gpuMem2, N * sizeof(int)) != hipSuccess) {
        std::cerr << "Failed to allocate GPU mem2" << std::endl;
	hipFree(gpuMem1);
	return -1;
    }
    if (hipMalloc((void**)&gpuResult, N * sizeof(int)) != hipSuccess) {
        std::cerr << "Failed to allocate GPU result memory" << std::endl;
	hipFree(gpuMem1);
	hipFree(gpuMem2);
	return -1;
    }

    // Fill the input array
    for (int i = 0; i < N; i++) {
        a[i] = i + 1;
	b[i] = N + i + 1;
    }

    if (hipMemcpy(gpuMem1, a, N * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        std::cerr << "Failed to copy a to gpuMem1" << std::endl;
	goto exit;
    }
    if (hipMemcpy(gpuMem2, b, N * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        std::cerr << "Failed to copy b to gpuMem2" << std::endl;
	goto exit;
    }

    // Add the arrays in GPU
    add<<<N, 1>>>(gpuMem1, gpuMem2, gpuResult);

    // Copy the output from GPU memory to CPU memory
    if (hipMemcpy(c, gpuResult, N * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
        std::cerr << "Failed to copy gpuResult to c" << std::endl;
	goto exit;
    }

    // Print the output
    for (int i = 0; i < N; i++) {
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    }

exit:
    hipFree(gpuMem1);
    hipFree(gpuMem2);
    hipFree(gpuResult);
    
    return 0;
}
